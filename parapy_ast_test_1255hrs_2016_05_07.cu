//Function foo parsed from ast_test.py


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define pi 3.14159265

__global__ void foo(float* arg,float* ret) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > 20) {
        return;
    }
    if (arg[index]>10) {
        ret[index]=1.0;
    }
    else {
        ret[index]=0.0;
    }
}

int main() {
    printf("STARTING MAIN FUNCTION\n");
    
    //Define constants to use
    const int N = 20;
    const int blocksize = 128;
    
    //Allocate the variables
    float arg [20]={1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20};
    float* ret=(float*)malloc(20*sizeof(float));
    
    //Declare and allocate the variables and copy it over to device
    float *argCuda;
    float *retCuda;
    const int csize = N*sizeof(float);
    hipMalloc( (void**)&argCuda, csize );
    hipMalloc( (void**)&retCuda, csize );
    hipMemcpy( argCuda, arg, csize, hipMemcpyHostToDevice );
    
    //Setup variables for cuda block and grid and then call function
    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( ((N + blocksize - 1) / blocksize), 1 );
    foo<<<dimGrid, dimBlock>>>(argCuda,retCuda);
    
    //Copy back result data
    hipMemcpy(ret, retCuda, N * sizeof(float), hipMemcpyDeviceToHost);
    
    //Free allocated memory
    hipFree( argCuda );
    hipFree( retCuda );
    free (ret);
    return 0;
}
