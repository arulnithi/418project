//Function foo parsed from ast_test.py


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define pi 3.14159265

__global__ void foo(int alpha,float* x,float* y,float* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > 20) {
        return;
    }
    result[index]=alpha*x[index]+y[index];
}

int main() {
    printf("STARTING MAIN FUNCTION\n");
    
    //Define constants to use
    const int N = 20;
    const int blocksize = 128;
    
    //Allocate the variables
    float x [20]={1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20};
    float y [20]={1, 2, 3, 4, 2, 4, 2, 63, 4, 7, 4, 8, 3, 1, 2, 5, 1, 5, 2, 2};
    float* result=(float*)malloc(20*sizeof(float));
    
    //Declare and allocate the variables and copy it over to device
    float *xCuda;
    float *yCuda;
    float *resultCuda;
    const int csize = N*sizeof(float);
    hipMalloc( (void**)&xCuda, csize );
    hipMalloc( (void**)&yCuda, csize );
    hipMalloc( (void**)&resultCuda, csize );
    hipMemcpy( xCuda, x, csize, hipMemcpyHostToDevice );
    hipMemcpy( yCuda, y, csize, hipMemcpyHostToDevice );
    
    //Setup variables for cuda block and grid and then call function
    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( ((N + blocksize - 1) / blocksize), 1 );
    foo<<<dimGrid, dimBlock>>>(2,xCuda,yCuda,resultCuda);
    
    //Copy back result data
    hipMemcpy(result, resultCuda, N * sizeof(float), hipMemcpyDeviceToHost);
    
    //Free allocated memory
    hipFree( xCuda );
    hipFree( yCuda );
    hipFree( resultCuda );
    free (result);
    return 0;
}
